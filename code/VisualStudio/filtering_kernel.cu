#include "hip/hip_runtime.h"
/*
	Non-separable 2D, 3D and 4D Filtering with CUDA
	Copyright (C) <2013>  Anders Eklund, andek034@gmail.com

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#ifndef FILTERING_KERNEL_CU_
#define FILTERING_KERNEL_CU_

#include "filtering.h"
#include "help_functions.cu"

__device__ __constant__ float c_Filter_2D[17][17];
__device__ __constant__ float c_Filter_3x3[3][3];
__device__ __constant__ float c_Filter_5x5[5][5];
__device__ __constant__ float c_Filter_7x7[7][7];
__device__ __constant__ float c_Filter_9x9[9][9];
__device__ __constant__ float c_Filter_11x11[11][11];
__device__ __constant__ float c_Filter_13x13[13][13];
__device__ __constant__ float c_Filter_15x15[15][15];
__device__ __constant__ float c_Filter_17x17[17][17];


__device__ __constant__ float c_Filter_3D[17][17][17];

__device__ __constant__ float c_Filter_3x3x3[3][3][3];
__device__ __constant__ float c_Filter_5x5x5[5][5][5];
__device__ __constant__ float c_Filter_7x7x7[7][7][7];
__device__ __constant__ float c_Filter_9x9x9[9][9][9];
__device__ __constant__ float c_Filter_11x11x11[11][11][11];
__device__ __constant__ float c_Filter_13x13x13[13][13][13];
__device__ __constant__ float c_Filter_15x15x15[15][15][15];
__device__ __constant__ float c_Filter_17x17x17[17][17][17];

__device__ float Conv_2D(float Image[64][96], int y, int x, int FILTER_W, int FILTER_H)
{
   float sum = 0.0f;

   int y_off = -(FILTER_H - 1)/2;
   for (int f_y = FILTER_H - 1; f_y >= 0; f_y--)
   {
      int x_off = -(FILTER_W - 1)/2;			
      for (int f_x = FILTER_W - 1; f_x >= 0; f_x--)
      {
         sum += Image[y + y_off][x + x_off] * c_Filter_2D[f_y][f_x];
         x_off++;
      }
      y_off++;
   }
	
   return sum;
}

__device__ float Conv_2D_Unrolled_7x7(float image[64][96], int y, int x)
{
	float pixel;
	float sum = 0.0f;
	
    pixel = image[y - 3][x - 3]; 
    sum += pixel * c_Filter_7x7[6][6];
 
    pixel = image[y - 2][x - 3]; 
    sum += pixel * c_Filter_7x7[5][6];

    pixel = image[y - 1][x - 3]; 
    sum += pixel * c_Filter_7x7[4][6];

    pixel = image[y + 0][x - 3]; 
    sum += pixel * c_Filter_7x7[3][6];

    pixel = image[y + 1][x - 3]; 
    sum += pixel * c_Filter_7x7[2][6];

    pixel = image[y + 2][x - 3]; 
    sum += pixel * c_Filter_7x7[1][6];

    pixel = image[y + 3][x - 3]; 
    sum += pixel * c_Filter_7x7[0][6];



    pixel = image[y - 3][x - 2]; 
    sum += pixel * c_Filter_7x7[6][5];

    pixel = image[y - 2][x - 2]; 
    sum += pixel * c_Filter_7x7[5][5];

    pixel = image[y - 1][x - 2]; 
    sum += pixel * c_Filter_7x7[4][5];

    pixel = image[y + 0][x - 2]; 
    sum += pixel * c_Filter_7x7[3][5];

    pixel = image[y + 1][x - 2]; 
    sum += pixel * c_Filter_7x7[2][5];

    pixel = image[y + 2][x - 2]; 
    sum += pixel * c_Filter_7x7[1][5];

    pixel = image[y + 3][x - 2]; 
    sum += pixel * c_Filter_7x7[0][5];


    pixel = image[y - 3][x - 1]; 
    sum += pixel * c_Filter_7x7[6][4];

    pixel = image[y - 2][x - 1]; 
    sum += pixel * c_Filter_7x7[5][4];

    pixel = image[y - 1][x - 1]; 
    sum += pixel * c_Filter_7x7[4][4];

    pixel = image[y + 0][x - 1]; 
    sum += pixel * c_Filter_7x7[3][4];

    pixel = image[y + 1][x - 1]; 
    sum += pixel * c_Filter_7x7[2][4];

    pixel = image[y + 2][x - 1]; 
    sum += pixel * c_Filter_7x7[1][4];

    pixel = image[y + 3][x - 1]; 
    sum += pixel * c_Filter_7x7[0][4];


    pixel = image[y - 3][x + 0]; 
    sum += pixel * c_Filter_7x7[6][3];

    pixel = image[y - 2][x + 0]; 
    sum += pixel * c_Filter_7x7[5][3];

    pixel = image[y - 1][x + 0]; 
    sum += pixel * c_Filter_7x7[4][3];

    pixel = image[y + 0][x + 0]; 
    sum += pixel * c_Filter_7x7[3][3];

    pixel = image[y + 1][x + 0]; 
    sum += pixel * c_Filter_7x7[2][3];

    pixel = image[y + 2][x + 0]; 
    sum += pixel * c_Filter_7x7[1][3];

    pixel = image[y + 3][x + 0]; 
    sum += pixel * c_Filter_7x7[0][3];

    pixel = image[y - 3][x + 1]; 
    sum += pixel * c_Filter_7x7[6][2];
  
    pixel = image[y - 2][x + 1]; 
    sum += pixel * c_Filter_7x7[5][2];
  
    pixel = image[y - 1][x + 1]; 
    sum += pixel * c_Filter_7x7[4][2];

    pixel = image[y + 0][x + 1]; 
    sum += pixel * c_Filter_7x7[3][2];

    pixel = image[y + 1][x + 1]; 
    sum += pixel * c_Filter_7x7[2][2];
 
    pixel = image[y + 2][x + 1]; 
    sum += pixel * c_Filter_7x7[1][2];

    pixel = image[y + 3][x + 1]; 
    sum += pixel * c_Filter_7x7[0][2];
 


    pixel = image[y - 3][x + 2]; 
    sum += pixel * c_Filter_7x7[6][1];

    pixel = image[y - 2][x + 2]; 
    sum += pixel * c_Filter_7x7[5][1];

    pixel = image[y - 1][x + 2]; 
    sum += pixel * c_Filter_7x7[4][1];
 
    pixel = image[y + 0][x + 2]; 
    sum += pixel * c_Filter_7x7[3][1];

    pixel = image[y + 1][x + 2]; 
    sum += pixel * c_Filter_7x7[2][1];
    
    pixel = image[y + 2][x + 2]; 
    sum += pixel * c_Filter_7x7[1][1];

    pixel = image[y + 3][x + 2]; 
    sum += pixel * c_Filter_7x7[0][1];


    pixel = image[y - 3][x + 3]; 
    sum += pixel * c_Filter_7x7[6][0];

    pixel = image[y - 2][x + 3]; 
    sum += pixel * c_Filter_7x7[5][0];

    pixel = image[y - 1][x + 3]; 
    sum += pixel * c_Filter_7x7[4][0];

    pixel = image[y + 0][x + 3]; 
    sum += pixel * c_Filter_7x7[3][0];

    pixel = image[y + 1][x + 3]; 
    sum += pixel * c_Filter_7x7[2][0];

    pixel = image[y + 2][x + 3]; 
    sum += pixel * c_Filter_7x7[1][0];
 
    pixel = image[y + 3][x + 3]; 
    sum += pixel * c_Filter_7x7[0][0];

	return sum;
}

/*
 This function performs non-separable 2D convolution by using texture memory.
*/

texture<float, 2, hipReadModeElementType> tex_Image;

__global__ void Convolution_2D_Texture(float* Filter_Response, int DATA_W, int DATA_H, int FILTER_W, int FILTER_H)
{
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;

   if (x >= DATA_W || y >= DATA_H)
        return;

   float sum = 0.0f;

   float y_off = -(FILTER_H - 1)/2 + 0.5f;
   for (int f_y = FILTER_H - 1; f_y >= 0; f_y--)
   {
        float x_off = -(FILTER_W - 1)/2 + 0.5f;			
        for (int f_x = FILTER_W - 1; f_x >= 0; f_x--)
        {
             sum += tex2D(tex_Image,x + x_off,y + y_off) * c_Filter_2D[f_y][f_x];
             x_off += 1.0f;
        }
        y_off += 1.0f;
    }

    Filter_Response[Get_2D_Index(x,y,DATA_W)] = sum;
}

__global__ void Convolution_2D_Texture_Unrolled_7x7(float* Filter_Response, int DATA_W, int DATA_H)
{
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;

   if (x >= DATA_W || y >= DATA_H)
        return;

   float sum = 0.0f;

   sum += tex2D(tex_Image, x - 3.0f + 0.5f, y - 3.0f + 0.5f) * c_Filter_7x7[6][6];
   sum += tex2D(tex_Image, x - 3.0f + 0.5f, y - 2.0f + 0.5f) * c_Filter_7x7[5][6];
   sum += tex2D(tex_Image, x - 3.0f + 0.5f, y - 1.0f + 0.5f) * c_Filter_7x7[4][6];
   sum += tex2D(tex_Image, x - 3.0f + 0.5f, y - 0.0f + 0.5f) * c_Filter_7x7[3][6];
   sum += tex2D(tex_Image, x - 3.0f + 0.5f, y + 1.0f + 0.5f) * c_Filter_7x7[2][6];
   sum += tex2D(tex_Image, x - 3.0f + 0.5f, y + 2.0f + 0.5f) * c_Filter_7x7[1][6];
   sum += tex2D(tex_Image, x - 3.0f + 0.5f, y + 3.0f + 0.5f) * c_Filter_7x7[0][6];
   
   sum += tex2D(tex_Image, x - 2.0f + 0.5f, y - 3.0f + 0.5f) * c_Filter_7x7[6][5];
   sum += tex2D(tex_Image, x - 2.0f + 0.5f, y - 2.0f + 0.5f) * c_Filter_7x7[5][5];
   sum += tex2D(tex_Image, x - 2.0f + 0.5f, y - 1.0f + 0.5f) * c_Filter_7x7[4][5];
   sum += tex2D(tex_Image, x - 2.0f + 0.5f, y - 0.0f + 0.5f) * c_Filter_7x7[3][5];
   sum += tex2D(tex_Image, x - 2.0f + 0.5f, y + 1.0f + 0.5f) * c_Filter_7x7[2][5];
   sum += tex2D(tex_Image, x - 2.0f + 0.5f, y + 2.0f + 0.5f) * c_Filter_7x7[1][5];
   sum += tex2D(tex_Image, x - 2.0f + 0.5f, y + 3.0f + 0.5f) * c_Filter_7x7[0][5];
   
   sum += tex2D(tex_Image, x - 1.0f + 0.5f, y - 3.0f + 0.5f) * c_Filter_7x7[6][4];
   sum += tex2D(tex_Image, x - 1.0f + 0.5f, y - 2.0f + 0.5f) * c_Filter_7x7[5][4];
   sum += tex2D(tex_Image, x - 1.0f + 0.5f, y - 1.0f + 0.5f) * c_Filter_7x7[4][4];
   sum += tex2D(tex_Image, x - 1.0f + 0.5f, y - 0.0f + 0.5f) * c_Filter_7x7[3][4];
   sum += tex2D(tex_Image, x - 1.0f + 0.5f, y + 1.0f + 0.5f) * c_Filter_7x7[2][4];
   sum += tex2D(tex_Image, x - 1.0f + 0.5f, y + 2.0f + 0.5f) * c_Filter_7x7[1][4];
   sum += tex2D(tex_Image, x - 1.0f + 0.5f, y + 3.0f + 0.5f) * c_Filter_7x7[0][4];
   
   sum += tex2D(tex_Image, x - 0.0f + 0.5f, y - 3.0f + 0.5f) * c_Filter_7x7[6][3];
   sum += tex2D(tex_Image, x - 0.0f + 0.5f, y - 2.0f + 0.5f) * c_Filter_7x7[5][3];
   sum += tex2D(tex_Image, x - 0.0f + 0.5f, y - 1.0f + 0.5f) * c_Filter_7x7[4][3];
   sum += tex2D(tex_Image, x - 0.0f + 0.5f, y - 0.0f + 0.5f) * c_Filter_7x7[3][3];
   sum += tex2D(tex_Image, x - 0.0f + 0.5f, y + 1.0f + 0.5f) * c_Filter_7x7[2][3];
   sum += tex2D(tex_Image, x - 0.0f + 0.5f, y + 2.0f + 0.5f) * c_Filter_7x7[1][3];
   sum += tex2D(tex_Image, x - 0.0f + 0.5f, y + 3.0f + 0.5f) * c_Filter_7x7[0][3];
   
   sum += tex2D(tex_Image, x + 1.0f + 0.5f, y - 3.0f + 0.5f) * c_Filter_7x7[6][2];
   sum += tex2D(tex_Image, x + 1.0f + 0.5f, y - 2.0f + 0.5f) * c_Filter_7x7[5][2];
   sum += tex2D(tex_Image, x + 1.0f + 0.5f, y - 1.0f + 0.5f) * c_Filter_7x7[4][2];
   sum += tex2D(tex_Image, x + 1.0f + 0.5f, y - 0.0f + 0.5f) * c_Filter_7x7[3][2];
   sum += tex2D(tex_Image, x + 1.0f + 0.5f, y + 1.0f + 0.5f) * c_Filter_7x7[2][2];
   sum += tex2D(tex_Image, x + 1.0f + 0.5f, y + 2.0f + 0.5f) * c_Filter_7x7[1][2];
   sum += tex2D(tex_Image, x + 1.0f + 0.5f, y + 3.0f + 0.5f) * c_Filter_7x7[0][2];
   
   sum += tex2D(tex_Image, x + 2.0f + 0.5f, y - 3.0f + 0.5f) * c_Filter_7x7[6][1];
   sum += tex2D(tex_Image, x + 2.0f + 0.5f, y - 2.0f + 0.5f) * c_Filter_7x7[5][1];
   sum += tex2D(tex_Image, x + 2.0f + 0.5f, y - 1.0f + 0.5f) * c_Filter_7x7[4][1];
   sum += tex2D(tex_Image, x + 2.0f + 0.5f, y - 0.0f + 0.5f) * c_Filter_7x7[3][1];
   sum += tex2D(tex_Image, x + 2.0f + 0.5f, y + 1.0f + 0.5f) * c_Filter_7x7[2][1];
   sum += tex2D(tex_Image, x + 2.0f + 0.5f, y + 2.0f + 0.5f) * c_Filter_7x7[1][1];
   sum += tex2D(tex_Image, x + 2.0f + 0.5f, y + 3.0f + 0.5f) * c_Filter_7x7[0][1];

   sum += tex2D(tex_Image, x + 3.0f + 0.5f, y - 3.0f + 0.5f) * c_Filter_7x7[6][0];
   sum += tex2D(tex_Image, x + 3.0f + 0.5f, y - 2.0f + 0.5f) * c_Filter_7x7[5][0];
   sum += tex2D(tex_Image, x + 3.0f + 0.5f, y - 1.0f + 0.5f) * c_Filter_7x7[4][0];
   sum += tex2D(tex_Image, x + 3.0f + 0.5f, y - 0.0f + 0.5f) * c_Filter_7x7[3][0];
   sum += tex2D(tex_Image, x + 3.0f + 0.5f, y + 1.0f + 0.5f) * c_Filter_7x7[2][0];
   sum += tex2D(tex_Image, x + 3.0f + 0.5f, y + 2.0f + 0.5f) * c_Filter_7x7[1][0];
   sum += tex2D(tex_Image, x + 3.0f + 0.5f, y + 3.0f + 0.5f) * c_Filter_7x7[0][0];

   Filter_Response[Get_2D_Index(x,y,DATA_W)] = sum;
}

/*
 This function performs non-separable 3D convolution by using texture memory.
*/

texture<float, 3, hipReadModeElementType> tex_Volume;


__global__ void Convolution_3D_Texture(float* Filter_Response, int DATA_W, int DATA_H, int DATA_D, int FILTER_W, int FILTER_H, int FILTER_D)
{
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;
   int z = blockIdx.z * blockDim.z + threadIdx.z;

   if (x >= DATA_W || y >= DATA_H || z >= DATA_D)
        return;

   float sum = 0.0f;

   float z_off = -(FILTER_D - 1)/2 + 0.5f;
   for (int f_z = FILTER_D - 1; f_z >= 0; f_z--)
   {
      float y_off = -(FILTER_H - 1)/2 + 0.5f;
      for (int f_y = FILTER_H - 1; f_y >= 0; f_y--)
      {
         float x_off = -(FILTER_W - 1)/2 + 0.5f;			
         for (int f_x = FILTER_W - 1; f_x >= 0; f_x--)
         {
            sum += tex3D(tex_Volume,x + x_off,y + y_off,z + z_off) * c_Filter_3D[f_y][f_x][f_z];
            x_off += 1.0f;
         }
         y_off += 1.0f;
     }
	 z_off += 1.0f;
   }

   Filter_Response[Get_3D_Index(x,y,z,DATA_W,DATA_H)] = sum;
}

__global__ void Convolution_3D_Texture_Unrolled_7x7x7(float* Filter_Response, int DATA_W, int DATA_H, int DATA_D)
{
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;
   int z = blockIdx.z * blockDim.z + threadIdx.z;

   if (x >= DATA_W || y >= DATA_H || z >= DATA_D)
        return;

   float sum = 0.0f;


   Filter_Response[Get_3D_Index(x,y,z,DATA_W,DATA_H)] = sum;
}

__global__ void Convolution_2D_Shared(float* Filter_Response, float* Image, int DATA_W, int DATA_H, int FILTER_W, int FILTER_H, int xBlockDifference, int yBlockDifference)
{
   int x = blockIdx.x * VALID_RESPONSES_X + threadIdx.x;
   int y = blockIdx.y * VALID_RESPONSES_Y + threadIdx.y;

   if ( (x >= (DATA_W + xBlockDifference)) || (y >= (DATA_H + yBlockDifference)) )
        return;

   __shared__ float s_Image[64][96]; // y, x

   // Reset shared memory
   s_Image[threadIdx.y][threadIdx.x]           = 0.0f;
   s_Image[threadIdx.y][threadIdx.x + 32]      = 0.0f;
   s_Image[threadIdx.y][threadIdx.x + 64]      = 0.0f;
   s_Image[threadIdx.y + 32][threadIdx.x]      = 0.0f;
   s_Image[threadIdx.y + 32][threadIdx.x + 32] = 0.0f;
   s_Image[threadIdx.y + 32][threadIdx.x + 64] = 0.0f;

   // Read data into shared memory

   if ( ((x-HALO) >= 0) && ((x-HALO) < DATA_W) && ((y-HALO) >= 0) && ((y-HALO) < DATA_H) )   
      s_Image[threadIdx.y][threadIdx.x] = Image[Get_2D_Index(x-HALO,y-HALO,DATA_W)];

   if ( ((x+32-HALO) < DATA_W) && ((y-HALO) >= 0) && ((y-HALO) < DATA_H) )
      s_Image[threadIdx.y][threadIdx.x + 32] = Image[Get_2D_Index(x+32-HALO,y-HALO,DATA_W)];

   if ( ((x+64-HALO) < DATA_W) && ((y-HALO) >= 0) && ((y-HALO) < DATA_H) ) 
      s_Image[threadIdx.y][threadIdx.x + 64] = Image[Get_2D_Index(x+64-HALO,y-HALO,DATA_W)];

   if ( ((x-HALO) >= 0) && ((x-HALO) < DATA_W) && ((y+32-HALO) < DATA_H) )
      s_Image[threadIdx.y + 32][threadIdx.x] = Image[Get_2D_Index(x-HALO,y+32-HALO,DATA_W)];

   if ( ((x+32-HALO) < DATA_W) && ((y+32-HALO) < DATA_H) )
      s_Image[threadIdx.y + 32][threadIdx.x + 32] = Image[Get_2D_Index(x+32-HALO,y+32-HALO, DATA_W)];

   if ( ((x+64-HALO) < DATA_W) && ((y+32-HALO) < DATA_H) )
      s_Image[threadIdx.y + 32][threadIdx.x + 64] = Image[Get_2D_Index(x+64-HALO,y+32-HALO,DATA_W)];
	
   __syncthreads();   

   // Only threads inside the image do the convolution

   if ( (x < DATA_W) && (y < DATA_H) )
      Filter_Response[Get_2D_Index(x,y,DATA_W)] = Conv_2D(s_Image,threadIdx.y+HALO,threadIdx.x+HALO,FILTER_H,FILTER_W);

   if ( ((x + 32) < DATA_W) && (y < DATA_H) )
      Filter_Response[Get_2D_Index(x+32,y,DATA_W)] = Conv_2D(s_Image,threadIdx.y+HALO,threadIdx.x+32+HALO,FILTER_H,FILTER_W);

   if (threadIdx.x < (32 - HALO*2))
   {
      if ( ((x + 64) < DATA_W) && (y < DATA_H) )
         Filter_Response[Get_2D_Index(x+64,y,DATA_W)] = Conv_2D(s_Image,threadIdx.y+HALO,threadIdx.x+64+HALO,FILTER_H,FILTER_W);
   }

   if (threadIdx.y < (32 - HALO*2))
   {
      if ( (x < DATA_W) && ((y + 32) < DATA_H) )
         Filter_Response[Get_2D_Index(x,y+32,DATA_W)] = Conv_2D(s_Image,threadIdx.y+32+HALO,threadIdx.x+HALO,FILTER_H,FILTER_W);
   }

   if (threadIdx.y < (32 - HALO*2))
   {
      if ( ((x + 32) < DATA_W) && ((y + 32) < DATA_H) )
         Filter_Response[Get_2D_Index(x+32,y+32,DATA_W)] = Conv_2D(s_Image,threadIdx.y+32+HALO,threadIdx.x+32+HALO,FILTER_H,FILTER_W);		
   } 

   if ( (threadIdx.x < (32 - HALO*2)) && (threadIdx.y < (32 - HALO*2)) )
   {
      if ( ((x + 64) < DATA_W) && ((y + 32) < DATA_H) )
         Filter_Response[Get_2D_Index(x+64,y+32,DATA_W)] = Conv_2D(s_Image,threadIdx.y+32+HALO,threadIdx.x+64+HALO,FILTER_H,FILTER_W);
   }

}

__global__ void Convolution_2D_Shared_Unrolled_7x7(float* Filter_Response, float* Image, int DATA_W, int DATA_H, int xBlockDifference, int yBlockDifference)
{
   int x = blockIdx.x * VALID_RESPONSES_X + threadIdx.x;
   int y = blockIdx.y * VALID_RESPONSES_Y + threadIdx.y;

   if ( (x >= (DATA_W + xBlockDifference)) || (y >= (DATA_H + yBlockDifference)) )
        return;

   __shared__ float s_Image[64][96]; // y, x

   // Reset shared memory
   s_Image[threadIdx.y][threadIdx.x]           = 0.0f;
   s_Image[threadIdx.y][threadIdx.x + 32]      = 0.0f;
   s_Image[threadIdx.y][threadIdx.x + 64]      = 0.0f;
   s_Image[threadIdx.y + 32][threadIdx.x]      = 0.0f;
   s_Image[threadIdx.y + 32][threadIdx.x + 32] = 0.0f;
   s_Image[threadIdx.y + 32][threadIdx.x + 64] = 0.0f;

   // Read data into shared memory

   if ( ((x-HALO) >= 0) && ((x-HALO) < DATA_W) && ((y-HALO) >= 0) && ((y-HALO) < DATA_H) )   
      s_Image[threadIdx.y][threadIdx.x] = Image[Get_2D_Index(x-HALO,y-HALO,DATA_W)];

   if ( ((x+32-HALO) < DATA_W) && ((y-HALO) >= 0) && ((y-HALO) < DATA_H) )
      s_Image[threadIdx.y][threadIdx.x + 32] = Image[Get_2D_Index(x+32-HALO,y-HALO,DATA_W)];

   if ( ((x+64-HALO) < DATA_W) && ((y-HALO) >= 0) && ((y-HALO) < DATA_H) ) 
      s_Image[threadIdx.y][threadIdx.x + 64] = Image[Get_2D_Index(x+64-HALO,y-HALO,DATA_W)];

   if ( ((x-HALO) >= 0) && ((x-HALO) < DATA_W) && ((y+32-HALO) < DATA_H) )
      s_Image[threadIdx.y + 32][threadIdx.x] = Image[Get_2D_Index(x-HALO,y+32-HALO,DATA_W)];

   if ( ((x+32-HALO) < DATA_W) && ((y+32-HALO) < DATA_H) )
      s_Image[threadIdx.y + 32][threadIdx.x + 32] = Image[Get_2D_Index(x+32-HALO,y+32-HALO, DATA_W)];

   if ( ((x+64-HALO) < DATA_W) && ((y+32-HALO) < DATA_H) )
      s_Image[threadIdx.y + 32][threadIdx.x + 64] = Image[Get_2D_Index(x+64-HALO,y+32-HALO,DATA_W)];
	
   __syncthreads();   

   // Only threads inside the image do the convolution

   if ( (x < DATA_W) && (y < DATA_H) )
      Filter_Response[Get_2D_Index(x,y,DATA_W)] = Conv_2D_Unrolled_7x7(s_Image,threadIdx.y+HALO,threadIdx.x+HALO);

   if ( ((x + 32) < DATA_W) && (y < DATA_H) )
      Filter_Response[Get_2D_Index(x+32,y,DATA_W)] = Conv_2D_Unrolled_7x7(s_Image,threadIdx.y+HALO,threadIdx.x+32+HALO);

   if (threadIdx.x < (32 - HALO*2))
   {
      if ( ((x + 64) < DATA_W) && (y < DATA_H) )
         Filter_Response[Get_2D_Index(x+64,y,DATA_W)] = Conv_2D_Unrolled_7x7(s_Image,threadIdx.y+HALO,threadIdx.x+64+HALO);
   }

   if (threadIdx.y < (32 - HALO*2))
   {
      if ( (x < DATA_W) && ((y + 32) < DATA_H) )
         Filter_Response[Get_2D_Index(x,y+32,DATA_W)] = Conv_2D_Unrolled_7x7(s_Image,threadIdx.y+32+HALO,threadIdx.x+HALO);
   }

   if (threadIdx.y < (32 - HALO*2))
   {
      if ( ((x + 32) < DATA_W) && ((y + 32) < DATA_H) )
         Filter_Response[Get_2D_Index(x+32,y+32,DATA_W)] = Conv_2D_Unrolled_7x7(s_Image,threadIdx.y+32+HALO,threadIdx.x+32+HALO);		
   } 

   if ( (threadIdx.x < (32 - HALO*2)) && (threadIdx.y < (32 - HALO*2)) )
   {
      if ( ((x + 64) < DATA_W) && ((y + 32) < DATA_H) )
         Filter_Response[Get_2D_Index(x+64,y+32,DATA_W)] = Conv_2D_Unrolled_7x7(s_Image,threadIdx.y+32+HALO,threadIdx.x+64+HALO);
   }

}

__global__ void Convolution_2D_Shared_For_3D(float* Filter_Response, float* Image, int z_offset, int DATA_W, int DATA_H, int DATA_D, int FILTER_W, int FILTER_H, int xBlockDifference, int yBlockDifference)
{
   int x = blockIdx.x * VALID_RESPONSES_X + threadIdx.x;
   int y = blockIdx.y * VALID_RESPONSES_Y + threadIdx.y;
   int z = blockIdx.z * blockDim.z + threadIdx.z;

   if ( (x >= (DATA_W + xBlockDifference)) || (y >= (DATA_H + yBlockDifference))  )
        return;

   __shared__ float s_Image[64][96]; // y, x

   // Reset shared memory
   s_Image[threadIdx.y][threadIdx.x]           = 0.0f;
   s_Image[threadIdx.y][threadIdx.x + 32]      = 0.0f;
   s_Image[threadIdx.y][threadIdx.x + 64]      = 0.0f;
   s_Image[threadIdx.y + 32][threadIdx.x]      = 0.0f;
   s_Image[threadIdx.y + 32][threadIdx.x + 32] = 0.0f;
   s_Image[threadIdx.y + 32][threadIdx.x + 64] = 0.0f;

   // Read data into shared memory

   if ( ((z + z_offset) >= 0) && ((z + z_offset) < DATA_D) )
   {
      if ( ((x-HALO) >= 0) && ((x-HALO) < DATA_W) && ((y-HALO) >= 0) && ((y-HALO) < DATA_H)  )   
         s_Image[threadIdx.y][threadIdx.x] = Image[Get_3D_Index(x-HALO,y-HALO,z+z_offset,DATA_W,DATA_H)];

      if ( ((x+32-HALO) < DATA_W) && ((y-HALO) >= 0) && ((y-HALO) < DATA_H)  )
         s_Image[threadIdx.y][threadIdx.x + 32] = Image[Get_3D_Index(x+32-HALO,y-HALO,z+z_offset,DATA_W,DATA_H)];

      if ( ((x+64-HALO) < DATA_W) && ((y-HALO) >= 0) && ((y-HALO) < DATA_H)  ) 
         s_Image[threadIdx.y][threadIdx.x + 64] = Image[Get_3D_Index(x+64-HALO,y-HALO,z+z_offset,DATA_W,DATA_H)];

      if ( ((x-HALO) >= 0) && ((x-HALO) < DATA_W) && ((y+32-HALO) < DATA_H)  )
         s_Image[threadIdx.y + 32][threadIdx.x] = Image[Get_3D_Index(x-HALO,y+32-HALO,z+z_offset,DATA_W,DATA_H)];

      if ( ((x+32-HALO) < DATA_W) && ((y+32-HALO) < DATA_H)  )
         s_Image[threadIdx.y + 32][threadIdx.x + 32] = Image[Get_3D_Index(x+32-HALO,y+32-HALO,z+z_offset,DATA_W,DATA_H)];

      if ( ((x+64-HALO) < DATA_W) && ((y+32-HALO) < DATA_H)  )
         s_Image[threadIdx.y + 32][threadIdx.x + 64] = Image[Get_3D_Index(x+64-HALO,y+32-HALO,z+z_offset,DATA_W,DATA_H)];
   }
	
   __syncthreads();   

   // Only threads inside the image do the convolution

   if ( (x < DATA_W) && (y < DATA_H) )
      Filter_Response[Get_3D_Index(x,y,z,DATA_W,DATA_H)] += Conv_2D(s_Image,threadIdx.y+HALO,threadIdx.x+HALO,FILTER_H,FILTER_W);

   if ( ((x + 32) < DATA_W) && (y < DATA_H) )
      Filter_Response[Get_3D_Index(x+32,y,z,DATA_W,DATA_H)] += Conv_2D(s_Image,threadIdx.y+HALO,threadIdx.x+32+HALO,FILTER_H,FILTER_W);

   if (threadIdx.x < (32 - HALO*2))
   {
      if ( ((x + 64) < DATA_W) && (y < DATA_H) )
         Filter_Response[Get_3D_Index(x+64,y,z,DATA_W,DATA_H)] += Conv_2D(s_Image,threadIdx.y+HALO,threadIdx.x+64+HALO,FILTER_H,FILTER_W);
   }

   if (threadIdx.y < (32 - HALO*2))
   {
      if ( (x < DATA_W) && ((y + 32) < DATA_H) )
         Filter_Response[Get_3D_Index(x,y+32,z,DATA_W,DATA_H)] += Conv_2D(s_Image,threadIdx.y+32+HALO,threadIdx.x+HALO,FILTER_H,FILTER_W);
   }

   if (threadIdx.y < (32 - HALO*2))
   {
      if ( ((x + 32) < DATA_W) && ((y + 32) < DATA_H) )
         Filter_Response[Get_3D_Index(x+32,y+32,z,DATA_W,DATA_H)] += Conv_2D(s_Image,threadIdx.y+32+HALO,threadIdx.x+32+HALO,FILTER_H,FILTER_W);		
   } 

   if ( (threadIdx.x < (32 - HALO*2)) && (threadIdx.y < (32 - HALO*2)) )
   {
      if ( ((x + 64) < DATA_W) && ((y + 32) < DATA_H) )
         Filter_Response[Get_3D_Index(x+64,y+32,z,DATA_W,DATA_H)] += Conv_2D(s_Image,threadIdx.y+32+HALO,threadIdx.x+64+HALO,FILTER_H,FILTER_W);
   }

}

__global__ void Convolution_2D_Shared_For_3D_Unrolled_7x7(float* Filter_Response, float* Image, int z_offset, int DATA_W, int DATA_H, int DATA_D, int xBlockDifference, int yBlockDifference)
{
   int x = blockIdx.x * VALID_RESPONSES_X + threadIdx.x;
   int y = blockIdx.y * VALID_RESPONSES_Y + threadIdx.y;
   int z = blockIdx.z * blockDim.z + threadIdx.z;

   if ( (x >= (DATA_W + xBlockDifference)) || (y >= (DATA_H + yBlockDifference))  )
        return;

   __shared__ float s_Image[64][96]; // y, x

   // Reset shared memory
   s_Image[threadIdx.y][threadIdx.x]           = 0.0f;
   s_Image[threadIdx.y][threadIdx.x + 32]      = 0.0f;
   s_Image[threadIdx.y][threadIdx.x + 64]      = 0.0f;
   s_Image[threadIdx.y + 32][threadIdx.x]      = 0.0f;
   s_Image[threadIdx.y + 32][threadIdx.x + 32] = 0.0f;
   s_Image[threadIdx.y + 32][threadIdx.x + 64] = 0.0f;

   // Read data into shared memory

   if ( ((z + z_offset) >= 0) && ((z + z_offset) < DATA_D) )
   {
      if ( ((x-HALO) >= 0) && ((x-HALO) < DATA_W) && ((y-HALO) >= 0) && ((y-HALO) < DATA_H)  )   
         s_Image[threadIdx.y][threadIdx.x] = Image[Get_3D_Index(x-HALO,y-HALO,z+z_offset,DATA_W,DATA_H)];

      if ( ((x+32-HALO) < DATA_W) && ((y-HALO) >= 0) && ((y-HALO) < DATA_H)  )
         s_Image[threadIdx.y][threadIdx.x + 32] = Image[Get_3D_Index(x+32-HALO,y-HALO,z+z_offset,DATA_W,DATA_H)];

      if ( ((x+64-HALO) < DATA_W) && ((y-HALO) >= 0) && ((y-HALO) < DATA_H)  ) 
         s_Image[threadIdx.y][threadIdx.x + 64] = Image[Get_3D_Index(x+64-HALO,y-HALO,z+z_offset,DATA_W,DATA_H)];

      if ( ((x-HALO) >= 0) && ((x-HALO) < DATA_W) && ((y+32-HALO) < DATA_H)  )
         s_Image[threadIdx.y + 32][threadIdx.x] = Image[Get_3D_Index(x-HALO,y+32-HALO,z+z_offset,DATA_W,DATA_H)];

      if ( ((x+32-HALO) < DATA_W) && ((y+32-HALO) < DATA_H)  )
         s_Image[threadIdx.y + 32][threadIdx.x + 32] = Image[Get_3D_Index(x+32-HALO,y+32-HALO,z+z_offset,DATA_W,DATA_H)];

      if ( ((x+64-HALO) < DATA_W) && ((y+32-HALO) < DATA_H)  )
         s_Image[threadIdx.y + 32][threadIdx.x + 64] = Image[Get_3D_Index(x+64-HALO,y+32-HALO,z+z_offset,DATA_W,DATA_H)];
   }
	
   __syncthreads();   

   // Only threads inside the image do the convolution

   if ( (x < DATA_W) && (y < DATA_H) )
      Filter_Response[Get_3D_Index(x,y,z,DATA_W,DATA_H)] += Conv_2D_Unrolled_7x7(s_Image,threadIdx.y+HALO,threadIdx.x+HALO);

   if ( ((x + 32) < DATA_W) && (y < DATA_H) )
      Filter_Response[Get_3D_Index(x+32,y,z,DATA_W,DATA_H)] += Conv_2D_Unrolled_7x7(s_Image,threadIdx.y+HALO,threadIdx.x+32+HALO);

   if (threadIdx.x < (32 - HALO*2))
   {
      if ( ((x + 64) < DATA_W) && (y < DATA_H) )
         Filter_Response[Get_3D_Index(x+64,y,z,DATA_W,DATA_H)] += Conv_2D_Unrolled_7x7(s_Image,threadIdx.y+HALO,threadIdx.x+64+HALO);
   }

   if (threadIdx.y < (32 - HALO*2))
   {
      if ( (x < DATA_W) && ((y + 32) < DATA_H) )
         Filter_Response[Get_3D_Index(x,y+32,z,DATA_W,DATA_H)] += Conv_2D_Unrolled_7x7(s_Image,threadIdx.y+32+HALO,threadIdx.x+HALO);
   }

   if (threadIdx.y < (32 - HALO*2))
   {
      if ( ((x + 32) < DATA_W) && ((y + 32) < DATA_H) )
         Filter_Response[Get_3D_Index(x+32,y+32,z,DATA_W,DATA_H)] += Conv_2D_Unrolled_7x7(s_Image,threadIdx.y+32+HALO,threadIdx.x+32+HALO);		
   } 

   if ( (threadIdx.x < (32 - HALO*2)) && (threadIdx.y < (32 - HALO*2)) )
   {
      if ( ((x + 64) < DATA_W) && ((y + 32) < DATA_H) )
         Filter_Response[Get_3D_Index(x+64,y+32,z,DATA_W,DATA_H)] += Conv_2D_Unrolled_7x7(s_Image,threadIdx.y+32+HALO,threadIdx.x+64+HALO);
   }
}


#endif

